#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__global__ void convolution2DBasicKernel(float *N, float *F, float *P, int r, int width, int height){
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    float Pvalue = 0.0f;

    for (int fRow = 0; fRow < 2*r + 1; ++fRow){
        for (int fCol = 0; fCol < 2*r + 1; ++fCol){
            int inRow = outRow - r + fRow;
            int inCol = outCol - r + fCol;
            if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width){
                Pvalue += F[fRow][fCol] * N[inRow * width + inCol];
            }
            
        }
    }
    P[outRow * width + outCol] = Pvalue;
}