#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define FILTER_RADIUS 2
#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2 * (FILTER_RADIUS))

__constant__ float F_c[2 * FILTER_RADIUS + 1][2 * FILTER_RADIUS + 1]

// To use this kernel, launch it with a block size whose dimensions match
// the input tile, but with a block count that is calculated from the output tile dimension.
__global__ void convolutionTiled2DConstMemKernel(float *N, float *P, int width, int height){
    int col = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
    int row = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;

    // Load input tile
    __shared__ N_s[IN_TILE_DIM][IN_TILE_DIM];
    if (row >= 0 && row < height && col >= 0 && col < width){
        N_s[threadIdx.y][threadIdx.x] = N[row * width + col];
    } else {
        N_s[threadIdx.y][threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Calculating output elements
    int tileCol = threadIdx.x - FILTER_RADIUS;
    int tileRow = threadIdx.y - FILTER_RADIUS;

    // Turning off the threads at the edge of the block
    if(row >= 0 && row < height && col >= 0 && col < width){
        if (tileCol >= 0 && tileCol < OUT_TILE_DIM && tileRow >= 0 && tileRow < OUT_TILE_DIM){
            float Pvalue = 0.0f;
            for (int fRow = 0 ; fRow < 2 * FILTER_RADIUS + 1; ++fRow){
                for (int fCol = 0; fCol < 2 * FILTER_RADIUS + 1; ++fCol){
                    Pvalue += F_c[fRow][fCol] * N_s[tileRow + fRow][tileCol + fCol];
                }
            }
            P[row * width + col] = Pvalue;
        }
    }
}